#include "algorithms.h"
#include "csv_reader.h"
#include "gpu_functions.h"

using namespace std;



void iris_dataset(){
  int rows, cols, rows_target, cols_target;                                       //reading data for IRIS dataset
  int K = 7;
  double** matrix = readCSV("data/iris_data.csv", rows, cols);  
  double** target = readCSV("data/iris_target.csv", rows_target, cols_target);  
  double* predicted_data;
  int *outputs;
  int mode;
  hipMallocManaged(&predicted_data, cols * sizeof(double));
  hipMallocManaged(&outputs, K * sizeof(int));
  double *distances;
  hipMallocManaged(&distances, rows * sizeof(double)); 

  for(int i=0; i<K; i++){
    outputs[i] = 0;
  }


  predicted_data[0] = 5;                                                          //some data to predict                  
  predicted_data[1] = 0.75;
  predicted_data[2] = 5;
  predicted_data[3] = 0.75; 
  size_t threads_per_block = 256;                                                   //execution configuration
  size_t number_of_blocks = (rows + threads_per_block - 1) / threads_per_block;
  
                                             
  auto start = std::chrono::steady_clock::now();                                        //GPU knn algorithm
  knn_algorithm_GPU<<<number_of_blocks, threads_per_block>>>(predicted_data, matrix, distances, rows, cols);
  hipDeviceSynchronize();
  mode =  predict(distances, target, 7, rows);
  std::cout << "Elapsed for GPU(microsec)=" << since<std::chrono::microseconds>(start).count() << std::endl;
  cout << "The predicted target for GPU is: " << mode << endl;


  start = std::chrono::steady_clock::now();                                                       //CPU knn algorithm
  knn_algorithm_CPU(predicted_data, matrix, distances, rows, cols);
  mode = predict(distances, target, 7, rows);
  std::cout << "Elapsed for CPU(microsec)=" << since<std::chrono::microseconds>(start).count() << std::endl;
  cout << "The predicted target for CPU is: " << mode << endl;


  for (int i = 0; i < rows; i++) {                            //free memory
    hipFree(&matrix[i]);  
  }
  hipFree(&matrix);  
  hipFree(&distances); 
  hipFree(&predicted_data);  
}




void mnist_dataset(){
  int rows_x_train, rows_y_train, rows_x_test, rows_y_test, cols, cols_target;                //reading MNIST dataset
  double** x_train = readCSV("data/x_train.csv", rows_x_train, cols);  
  double** y_train = readCSV("data/y_train.csv", rows_y_train, cols_target);  
  double* distances;
  double** x_test = readCSV("data/x_train.csv", rows_x_test, cols);  
  double** y_test = readCSV("data/y_train.csv", rows_y_test, cols_target); 
  double* predictable_sample;
  int correct_prediction = 0;
  hipMallocManaged(&distances, rows_x_train * sizeof(double));    
  hipMallocManaged(&predictable_sample, cols * sizeof(double));

  size_t threads_per_block = 1024;                                                 
  size_t number_of_blocks = (rows_x_train + threads_per_block - 1) / threads_per_block;


   //GPU knn algorithm
  int samples_number = 500;
  auto start = std::chrono::steady_clock::now();                
  for(int i= 0; i<samples_number; i++){
    predictable_sample = x_test[i];
    knn_algorithm_GPU<<<number_of_blocks, threads_per_block>>>(predictable_sample, x_train, distances, rows_x_train, cols);
    hipDeviceSynchronize();

    if(predict(distances, y_train, 7, rows_x_train) == y_test[i][0]){
      correct_prediction++;
    }
    
  }
  std::cout << "Elapsed for GPU =  " << since<std::chrono::microseconds>(start).count() * 1e-6 << "  s" <<std::endl;
  std::cout << "Accuracy =  " << (double)correct_prediction/samples_number * 100 << "%" << std::endl;
  

 //CPU knn algorithm
  correct_prediction = 0;
  start = std::chrono::steady_clock::now();
  for(int i= 0; i<samples_number; i++){
    predictable_sample = x_test[i];
    knn_algorithm_CPU(predictable_sample, x_train, distances, rows_x_train, cols);
      if(predict(distances, y_train, 7, rows_x_train) == y_test[i][0]){
        correct_prediction++;
      }
    
  }
  std::cout << "Elapsed for CPU =  " << since<std::chrono::microseconds>(start).count() * 1e-6 << "  s" <<std::endl;
  std::cout << "Accuracy =  " << (double)correct_prediction / samples_number * 100 << "%" << std::endl;
 


}

int main(){
  //two datasets to test

  iris_dataset();           
  // mnist_dataset();

}
